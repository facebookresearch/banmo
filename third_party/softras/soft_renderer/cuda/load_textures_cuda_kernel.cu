#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace {
template <typename scalar_t>
__global__ void load_textures_cuda_kernel(
    const scalar_t* __restrict__ image,
    const scalar_t* __restrict__ faces,
    const int32_t* __restrict__ is_update,
    scalar_t* __restrict__ textures, 
    size_t texture_size,
    size_t texture_res,
    size_t image_height,
    size_t image_width) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i * 3 >= texture_size) {
      return;
  }
  const int R = texture_res;
  const int fn = i / (R * R);
  const int w_y = (i % (R * R)) / R;
  const int w_x = i % R;
  // compute barycentric coordinate
  scalar_t w0, w1, w2;
  if (w_x + w_y < R) {
      w0 = (w_x + 1. / 3.) / R;
      w1 = (w_y + 1. / 3.) / R;
      w2 = 1. - w0 - w1;
  } else {
      w0 = ((R - 1. - w_x) + 2. / 3.) / R;
      w1 = ((R - 1. - w_y) + 2. / 3.) / R;
      w2 = 1. - w0 - w1;
  }
  const scalar_t* face = &faces[fn * 3 * 2];
  scalar_t* texture = &textures[i * 3];
  if (is_update[fn] == 0) return;
  
  const scalar_t pos_x = (
      (face[2 * 0 + 0] * w0 + face[2 * 1 + 0] * w1 + face[2 * 2 + 0] * w2) * (image_width - 1));
  const scalar_t pos_y = (
      (face[2 * 0 + 1] * w0 + face[2 * 1 + 1] * w1 + face[2 * 2 + 1] * w2) * (image_height - 1));
  if (1) {
      /* bilinear sampling */
      const scalar_t weight_x1 = pos_x - (int)pos_x;
      const scalar_t weight_x0 = 1 - weight_x1;
      const scalar_t weight_y1 = pos_y - (int)pos_y;
      const scalar_t weight_y0 = 1 - weight_y1;
      for (int k = 0; k < 3; k++) {
          scalar_t c = 0;
          c += image[((int)pos_y * image_width + (int)pos_x) * 3 + k] * (weight_x0 * weight_y0);
          c += image[((int)(pos_y + 1) * image_width + (int)pos_x) * 3 + k] * (weight_x0 * weight_y1);
          c += image[((int)pos_y * image_width + ((int)pos_x) + 1) * 3 + k] * (weight_x1 * weight_y0);
          c += image[((int)(pos_y + 1)* image_width + ((int)pos_x) + 1) * 3 + k] * (weight_x1 * weight_y1);
          texture[k] = c;
      }
  } else {
      /* nearest neighbor */
      const int pos_xi = round(pos_x);
      const int pos_yi = round(pos_y);
      for (int k = 0; k < 3; k++) {
          texture[k] = image[(pos_yi * image_width + pos_xi) * 3 + k];
      }
  }
}
}

at::Tensor load_textures_cuda(
        at::Tensor image,
        at::Tensor faces,
        at::Tensor textures,
        at::Tensor is_update) {
    // texture_size = size of the textures tensor
    const auto texture_size = textures.numel();
    // notice that texture_res != texture_res
    const auto texture_res = sqrt(textures.size(1));
    const auto image_height = image.size(0);
    const auto image_width = image.size(1);
    
    const int threads = 1024;
    const dim3 blocks ((texture_size / 3 - 1) / threads + 1);

    AT_DISPATCH_FLOATING_TYPES(image.type(), "load_textures_cuda", ([&] {
      load_textures_cuda_kernel<scalar_t><<<blocks, threads>>>(
          image.data<scalar_t>(),
          faces.data<scalar_t>(),
          is_update.data<int32_t>(),
          textures.data<scalar_t>(),
          texture_size,
          texture_res,
          image_height,
          image_width);
      }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
            printf("Error in load_textures: %s\n", hipGetErrorString(err));
    return textures;
}
